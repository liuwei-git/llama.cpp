#include "hip/hip_runtime.h"
#include "scale.cuh"

static __global__ void scale_f32(const float * x, float * dst, const float scale, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = scale * x[i];
}

static void scale_f32_cuda(const float * x, float * dst, const float scale, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SCALE_BLOCK_SIZE - 1) / CUDA_SCALE_BLOCK_SIZE;
    scale_f32<<<num_blocks, CUDA_SCALE_BLOCK_SIZE, 0, stream>>>(x, dst, scale, k);
}

void ggml_cuda_op_scale(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    float scale;
    memcpy(&scale, dst->op_params, sizeof(float));

    scale_f32_cuda(src0_d, dst_d, scale, ggml_nelements(src0), stream);
    CUDA_CHECK(hipGetLastError());
}

// ggml_map_custom1
struct ggml_map_custom1_op_params {
    ggml_custom1_op_t fun;
    int n_tasks;
    void * userdata;
};

void ggml_cuda_op_map_custom1(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    
    const struct ggml_tensor * a = dst->src[0];

    struct ggml_map_custom1_op_params p;
    memcpy(&p, dst->op_params, sizeof(p));

    size_t size = a->nb[0] * a->ne[0] * a->ne[1] * a->ne[2] * a->ne[3];
    char * data = new char[size];

    CUDA_CHECK(hipMemcpyAsync(data, (const char *)a->data, size, hipMemcpyDeviceToHost, ctx.stream()));

    p.fun(dst, a, 0, 0, data);
    delete data;

    CUDA_CHECK(hipGetLastError());
}